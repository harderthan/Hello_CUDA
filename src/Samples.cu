// Harderthan, kheo1772@gmail.com
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// kernelFunction_01
__global__ void kernel_01(void){

}
void kernelFunction_01(void){
    kernel_01<<<1,1>>>();
}

// kernelFunction_02
__global__ void kernel_02(int a, int b, int *c){
    *c = a + b;
}
void kernelFunction_02(void){
    int c;
    int *dev_c;

    HANDLE_ERROR( hipMalloc( (void**) &dev_c, sizeof(int) ) );

    kernel_02<<<1,1>>>(2,7,dev_c);

    HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost) );
    printf( "2 + 7 = %d \n", c);
    hipFree( dev_c );

    return;
}

// kernelFunction_03
static int POINTS_HEIGHT = 32;
static int POINTS_WIDTH = 1;
static int POINTS_NUM = POINTS_WIDTH * POINTS_HEIGHT;
static int POINTS_STEP = 3;

__global__ void kernel_03(float *src_dev_points, float *dst_dev_points, int points_num, int step_size){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < points_num){
		dst_dev_points[idx * step_size + 0] = src_dev_points[idx * step_size + 0] * 10;
		dst_dev_points[idx * step_size + 1] = src_dev_points[idx * step_size + 1] * 10;
		dst_dev_points[idx * step_size + 2] = src_dev_points[idx * step_size + 2] * 10;
	}
}
void kernelFunction_03(){
    float *src_points;
    float *dst_points;
    src_points = (float *) malloc(sizeof(float) * POINTS_NUM * POINTS_STEP);
    dst_points = (float *) malloc(sizeof(float) * POINTS_NUM * POINTS_STEP);

    for(int idx = 0; idx < POINTS_NUM; ++idx){
    	src_points[idx * POINTS_STEP + 0] = (float) rand();
    	src_points[idx * POINTS_STEP + 1] = (float) rand();
    	src_points[idx * POINTS_STEP + 2] = (float) rand();
    }

    {
    	    float *src_dev_points;
    	    float *dst_dev_points;
    	    HANDLE_ERROR( hipMalloc( (void**) &src_dev_points, sizeof(float) * POINTS_NUM * POINTS_STEP ) );
    	    HANDLE_ERROR( hipMalloc( (void**) &dst_dev_points, sizeof(float) * POINTS_NUM * POINTS_STEP ) );

    	    HANDLE_ERROR( hipMemcpy( src_dev_points, src_points, sizeof(float) * POINTS_NUM * POINTS_STEP, hipMemcpyHostToDevice));

    	    int thread_size = POINTS_HEIGHT;
    	    int block_size = (POINTS_HEIGHT * POINTS_WIDTH + POINTS_HEIGHT - 1 ) / POINTS_HEIGHT;
    	    kernel_03<<<block_size,thread_size>>>(src_dev_points, dst_dev_points, POINTS_NUM, POINTS_STEP);

    	    HANDLE_ERROR( hipMemcpy( dst_points, dst_dev_points, sizeof(float) * POINTS_NUM * POINTS_STEP, hipMemcpyDeviceToHost));

    	    for(int idx = 0; idx < POINTS_NUM; ++idx){
    	    	printf("id: %d\n", idx);
    	    	printf("%f", src_points[idx * POINTS_STEP + 0]);
    	    	printf(", %f", src_points[idx * POINTS_STEP + 1]);
    	    	printf(", %f \n", src_points[idx * POINTS_STEP + 2]);
    	        printf("%f", dst_points[idx * POINTS_STEP + 0]);
    	        printf(", %f", dst_points[idx * POINTS_STEP + 1]);
    	        printf(", %f \n", dst_points[idx * POINTS_STEP + 2]);
    	    }

    	    hipFree( src_dev_points );
    	    hipFree( dst_dev_points );
    }
}
